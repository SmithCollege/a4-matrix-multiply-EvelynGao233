#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void matrixMultiCuBLAS(float *M, float *N, float *P, int Width) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasHandle_t handle;

    hipblasCreate(&handle);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Width, Width, Width, &alpha, M, Width, N, Width, &beta, P, Width);

    hipblasDestroy(handle);
}

int main() {
    int sizes[] = {128, 256, 512, 1024};

    for (int s = 0; s < 4; s++) {
        int Width = sizes[s];
        int size = Width * Width * sizeof(float);
        float *M, *N, *P;

        hipMallocManaged(&M, size);
        hipMallocManaged(&N, size);
        hipMallocManaged(&P, size);

        // Initialize M and N
        for (int i = 0; i < Width * Width; i++) {
            M[i] = 1.0f;
            N[i] = 2.0f;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        matrixMultiCuBLAS(M, N, P, Width);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Print time 
        printf("Matrix size: %d x %d, Time taken (cuBLAS): %f ms\n", Width, Width, milliseconds);

        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipFree(M);
        hipFree(N);
        hipFree(P);

        printf("Matrix multiplication on cuBLAS for size %d completed.\n\n", Width);
    }

    return 0;
}
